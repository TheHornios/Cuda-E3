#include "hip/hip_runtime.h"
﻿/**
* ARQUITECTURA DE COMPUTADORES
* 2º Grado en Ingenieria Informatica
*
* Entrega 3
*
* Alumno: Rodrigo Pascual Arnaiz y Villar Solla, Alejandro
* Fecha: 30/11/2022
*
*/

///////////////////////////////////////////////////////////////////////////
// includes
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "./gpu_bitmap.h"

///////////////////////////////////////////////////////////////////////////
// prototipos

__host__ void leerBMP_RGBA(const char* nombre, int* w, int* h, unsigned char** imagen);
__host__ int mcd(int x, int y);
__global__ void convertirRgbBancoyNegro(unsigned char* colores);

///////////////////////////////////////////////////////////////////////////

// MAIN: rutina principal ejecutada en el host
int main(int argc, char** argv)
{

	// Declaramos las variables
	unsigned char* host_bitmap, * dev_bitmap;
	int ancho, alto;
	int thread_x_tam, thread_y_tam, divisor;
	float elapsedTime;


	// Leemos la imagen con la funcion proporcionada en los apuntes 
	leerBMP_RGBA("imagen.bmp", &ancho, &alto, &host_bitmap);


	// Obtener utilizando el MCD el tamañlo que van a tener los hilos 
	divisor = mcd(ancho, alto);
	thread_x_tam = ancho / divisor;
	thread_y_tam = alto / divisor;

	// Declaracion del bitmap:
	// Inicializacion de la estructura RenderGPU
	RenderGPU foto(ancho, alto);

	// Tamaño del bitmap en bytes
	size_t size = foto.image_size();

	// Asignacion y reserva de la memoria en el host (framebuffer) 
	unsigned char* host_imagen = foto.get_ptr();

	// Reservamos el hueco del dev bitmap
	hipMalloc((void**)&dev_bitmap, size);


	// Movemos el bitmap del host a device 
	hipMemcpy(dev_bitmap, host_bitmap, size, hipMemcpyHostToDevice);

	// Se calula el numero de bloques que se va a necesitar por cada hilo, para ello 
	// dividimos el ancho o el alto entre el tamaño de los hilos que le corresponda 
	dim3 Nbloques(ancho / thread_x_tam, alto / thread_y_tam);

	// Definimos los hilos teniendo ecuenta los tamaños calculados antes para cada hilo
	// usamos la variable thread_x_tam, thread_y_tam
	dim3 hilosB(thread_x_tam, thread_y_tam);

	// Inicializamos los eventos 
	hipEvent_t inicio, fin;
	hipEventCreate(&inicio);
	hipEventCreate(&fin);

	// El evento de inicio lo ponemos a 0
	hipEventRecord(inicio, 0);

	// Lanzamos la función kernel que va a convertir todos los pixeles a banco y negro
	convertirRgbBancoyNegro << <Nbloques, hilosB >> > (dev_bitmap);;

	// Registramos el evento de FIN como 0 y sincronizamos los eventos 
	hipEventRecord(fin, 0);
	hipEventSynchronize(fin);

	// Recogemos el bitmap desde la GPU para visualizarlo
	hipMemcpy(host_imagen, dev_bitmap, size, hipMemcpyDeviceToHost);

	// Calculamos la elipsis de tiempo transcurrido
	hipEventElapsedTime(&elapsedTime, inicio, fin);

	// visualizamos el tamaño del kernel 
	printf("\nTamño del kernel: " );
	printf("\nTamño numero de bloques en x:	%i -> con %i hilos", Nbloques.x, ( hilosB.x * hilosB.y) );
	printf("\nTamño numero de bloques en y:	%i -> con %i hilos", Nbloques.y, (hilosB.x * hilosB.y));
	printf("\nTamño numero de hilos en x:	%i", hilosB.x);
	printf("\nTamño numero de hilos en y:	%i", hilosB.y);
	printf("\nTotal de: %i",  (hilosB.x * hilosB.y) * Nbloques.x  * Nbloques.y );


	// Visualizacion y salida
	printf("\nEl tiempo de procesamiento ha sido de %f ms", elapsedTime);

	printf("\n...pulsa [ESC] para finalizar...");


	// Destruimos todos los eventos 
	hipEventDestroy(inicio);
	hipEventDestroy(fin);

	// Visualización de la foto pasada a negro 
	foto.display_and_exit();


	// Fin del programa
	return 0;
}

/**
* Funcion: leerBMP_RGBA ( HOST )
* Objetivo: Función que se encarga de leer un archivo de BMP
*
* @param const char* nombre -> Nombre del BMP
* @param int* w -> Ancho de la imagen en pixeles
* @param int* h -> Alto de la imagen en pixeles
* @param unsigned char** imagen -> Puntero al array de datos de la imagen en formato RGBA
* @return: void
*/
__host__ void leerBMP_RGBA(const char* nombre, int* w, int* h, unsigned char** imagen)
{
	// Lectura del archivo .BMP
	FILE* archivo;

	// Abrimos el archivo en modo solo lectura binaria
	if ((archivo = fopen(nombre, "rb")) == NULL)
	{
		printf("\nERROR ABRIENDO EL ARCHIVO %s...", nombre);
		// salida
		printf("\npulsa [INTRO] para finalizar");
		getchar();
		exit(1);
	}
	printf("> Archivo [%s] abierto:\n", nombre);

	// En Windows, la cabecera tiene un tamaño de 54 bytes:
	// 14 bytes (BMP header) + 40 bytes (DIB header)
	// BMP HEADER
	// Extraemos cada campo y lo almacenamos en una variable del tipo adecuado
	// posición 0x00 -> Tipo de archivo: "BM" (leemos 2 bytes)
	unsigned char tipo[2];
	fread(tipo, 1, 2, archivo);

	// Comprobamos que es un archivo BMP
	if (tipo[0] != 'B' || tipo[1] != 'M')
	{
		printf("\nERROR: EL ARCHIVO %s NO ES DE TIPO BMP...", nombre);
		// salida
		printf("\npulsa [INTRO] para finalizar");
		getchar();
		exit(1);
	}

	// posición 0x02 -> Tamaño del archivo .bmp (leemos 4 bytes)
	unsigned int file_size;
	fread(&file_size, 4, 1, archivo);

	// posición 0x06 -> Campo reservado (leemos 2 bytes)
	// posición 0x08 -> Campo reservado (leemos 2 bytes)
	unsigned char buffer[4];
	fread(buffer, 1, 4, archivo);

	// posición 0x0A -> Offset a los datos de imagen (leemos 4 bytes)
	unsigned int offset;
	fread(&offset, 4, 1, archivo);

	// imprimimos los datos
	printf(" \nDatos de la cabecera BMP\n");
	printf("> Tipo de archivo : %c%c\n", tipo[0], tipo[1]);
	printf("> Tamano del archivo : %u KiB\n", file_size / 1024);
	printf("> Offset de datos : %u bytes\n", offset);

	// DIB HEADER
	// Extraemos cada campo y lo almacenamos en una variable del tipo adecuado
	// posición 0x0E -> Tamaño de la cabecera DIB (BITMAPINFOHEADER) (leemos 4bytes)
	unsigned int header_size;
	fread(&header_size, 4, 1, archivo);

	// posición 0x12 -> Ancho de la imagen (leemos 4 bytes)
	unsigned int ancho;
	fread(&ancho, 4, 1, archivo);

	// posición 0x16 -> Alto de la imagen (leemos 4 bytes)
	unsigned int alto;
	fread(&alto, 4, 1, archivo);

	// posición 0x1A -> Numero de planos de color (leemos 2 bytes)
	unsigned short int planos;
	fread(&planos, 2, 1, archivo);

	// posición 0x1C -> Profundidad de color (leemos 2 bytes)
	unsigned short int color_depth;
	fread(&color_depth, 2, 1, archivo);

	// posicion 0x1E -> Tipo de compresión (leemos 4 bytes)
	unsigned int compresion;
	fread(&compresion, 4, 1, archivo);

	// imprimimos los datos
	printf(" \nDatos de la cabecera DIB\n");
	printf("> Tamano de la cabecera: %u bytes\n", header_size);
	printf("> Ancho de la imagen : %u pixeles\n", ancho);
	printf("> Alto de la imagen : %u pixeles\n", alto);
	printf("> Planos de color : %u\n", planos);
	printf("> Profundidad de color : %u bits/pixel\n", color_depth);
	printf("> Tipo de compresion : %s\n", (compresion == 0) ? "none" : "unknown");

	// LEEMOS LOS DATOS DEL ARCHIVO
	// Calculamos espacio para una imagen de tipo RGBA:
	size_t img_size = ancho * alto * 4;

	// Reserva para almacenar los datos del bitmap
	unsigned char* datos = (unsigned char*)malloc(img_size);

	// Desplazamos el puntero FILE hasta el comienzo de los datos de imagen: 0 +offset
	fseek(archivo, offset, SEEK_SET);

	// Leemos píxel a pixel, reordenamos (BGR -> RGB) e insertamos canal alfa
	unsigned int pixel_size = color_depth / 8;
	for (unsigned int i = 0; i < ancho * alto; i++)
	{
		fread(buffer, 1, pixel_size, archivo); // leemos el pixel i
		datos[i * 4 + 0] = buffer[2]; // escribimos canal R
		datos[i * 4 + 1] = buffer[1]; // escribimos canal G
		datos[i * 4 + 2] = buffer[0]; // escribimos canal B
		datos[i * 4 + 3] = buffer[3]; // escribimos canal alfa (si lo hay)
	}
	// Cerramos el archivo
	fclose(archivo);

	// PARAMETROS DE SALIDA
	// Ancho de la imagen en pixeles
	*w = ancho;

	// Alto de la imagen en pixeles
	*h = alto;

	// Puntero al array de datos RGBA
	*imagen = datos;

	// Salida
	return;
}

/**
* Funcion: convertirRgbBancoyNegro ( GLOBAL )
* Objetivo: Función que convierte una matriz de colores RGB a
*	blanco y negro, formula:
*	Y = 0.299×R + 0.587×G + 0.114×B
*
* @param unsigned char *colores -> array de colores
* @return: void
*/
__global__ void convertirRgbBancoyNegro(unsigned char* imagen)
{
	// coordenada vertical de cada hilo
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	// coordenada horizontal de cada hilo
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	// coordenada global de cada hilo (indice para acceder a la memoria)
	int pos = x + y * blockDim.x * gridDim.x;

	// cada hilo obtiene la posicion de un pixel
	int pixel = pos * 4;

	// Calculamos el tono de gris con la fórmula
	int Y = 0.299F * imagen[pixel + 0] + 0.587F * imagen[pixel + 1] + 0.114F * imagen[pixel + 2];

	// Establecemos los nuevos valores de color en el RGB
	imagen[pixel + 0] = Y;
	imagen[pixel + 1] = Y;
	imagen[pixel + 2] = Y;

}

/**
* Función que retorna el maximo comun divisor 
*
* @param int x -> primner valor
* @param int 2 -> segundo valor 
* @return int -> resultado 
*/
__host__ int mcd(int x, int y)
{
	return y ? mcd(y, x % y) : x;
	 
}